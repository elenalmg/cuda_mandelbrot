#include <vector>
#include <hip/hip_runtime.h>

__device__ int mandelbrot_escape_time(float x, float y, int max_iter, float escape_radius_squared) {
    float zr = 0.0f, zi = 0.0f;
    float zr2 = 0.0f, zi2 = 0.0f;
    int iter = 0;

    while (zr2 + zi2 < escape_radius_squared && iter < max_iter) {
        zi = 2.0f * zr * zi + y;
        zr = zr2 - zi2 + x;
        
        zr2 = zr * zr;
        zi2 = zi * zi;

        iter++;
    }

    if (zr2 + zi2 >= escape_radius_squared) {
        return iter;
    } else {
        return -1;
    }
}

__device__ float mandelbrot_smooth_color(float x, float y, int max_iter, float escape_radius_squared) {
    float zr = 0.0f, zi = 0.0f;
    float zr2 = 0.0f, zi2 = 0.0f;
    int iter = 0;

    while (zr2 + zi2 < escape_radius_squared && iter < max_iter) {
        zi = 2.0f * zr * zi + y;
        zr = zr2 - zi2 + x;
        
        zr2 = zr * zr;
        zi2 = zi * zi;

        iter++;
    }

    if (iter < max_iter) {
        float log_zn = logf(zr2 + zi2) / 2.0f;
        float nu = logf(log_zn / logf(2.0f)) / logf(2.0f);
        return iter + 1 - nu;
    } else {
        return max_iter;
    }
}

__global__ void mandelbrot_kernel(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius_squared, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < width && idy < height) {
        float pixel_size_x = (x_max - x_min) / width;
        float pixel_size_y = (y_max - y_min) / height;
        float real = x_min + idx * pixel_size_x;
        float imag = y_min + idy * pixel_size_y;

        results[idy * width + idx] = mandelbrot_escape_time(real, imag, max_iter, escape_radius_squared);
    }
}

__global__ void mandelbrot_kernel_smooth(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius_squared, float* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < width && idy < height) {
        float pixel_size_x = (x_max - x_min) / width;
        float pixel_size_y = (y_max - y_min) / height;
        float real = x_min + idx * pixel_size_x;
        float imag = y_min + idy * pixel_size_y;

        results[idy * width + idx] = mandelbrot_smooth_color(real, imag, max_iter, escape_radius_squared);
    }
}

extern "C" void compute_grid_cuda(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius_squared, int* results) {
    int* d_results;
    size_t size = width * height * sizeof(int);
    hipMalloc(&d_results, size);
    
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x, (height + threads_per_block.y - 1) / threads_per_block.y);
    
    mandelbrot_kernel<<<num_blocks, threads_per_block>>>(x_min, y_min, x_max, y_max, width, height, max_iter, escape_radius_squared, d_results);
    
    hipMemcpy(results, d_results, size, hipMemcpyDeviceToHost);
    hipFree(d_results);
}

extern "C" void compute_grid_cuda_smooth(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius_squared, float* results) {
    float* d_results;
    size_t size = width * height * sizeof(float);
    hipMalloc(&d_results, size);
    
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x, (height + threads_per_block.y - 1) / threads_per_block.y);
    
    mandelbrot_kernel_smooth<<<num_blocks, threads_per_block>>>(x_min, y_min, x_max, y_max, width, height, max_iter, escape_radius_squared, d_results);
    
    hipMemcpy(results, d_results, size, hipMemcpyDeviceToHost);
    hipFree(d_results);
}
