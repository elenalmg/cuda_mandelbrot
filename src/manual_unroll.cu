#include <vector>
#include <hip/hip_runtime.h>

__device__ int mandelbrot_escape_time_manual_unroll(float x, float y, int max_iter, float escape_radius_squared) {
    float zr = x, zi = y;
    float zr2 = zr * zr, zi2 = zi * zi;
    int iter = 0;

    while (zr2 + zi2 < escape_radius_squared && iter < max_iter) {
        float new_zr = zr2 - zi2 + x;
        float new_zi = 2.0f * zr * zi + y;

        zr = new_zr * new_zr - new_zi * new_zi + x;
        zi = 2.0f * new_zr * new_zi + y;

        zr2 = zr * zr;
        zi2 = zi * zi;

        iter += 2; // Two iterations per loop
    }

    if (zr2 + zi2 >= escape_radius_squared) {
        return iter;
    } else {
        return -1;
    }
}

__device__ float mandelbrot_smooth_color_manual_unroll(float x, float y, int max_iter, float escape_radius_squared) {
    float zr = x, zi = y;
    float zr2 = zr * zr, zi2 = zi * zi;
    int iter = 0;

    while (zr2 + zi2 < escape_radius_squared && iter < max_iter) {
        float new_zr = zr2 - zi2 + x;
        float new_zi = 2.0f * zr * zi + y;

        zr = new_zr * new_zr - new_zi * new_zi + x;
        zi = 2.0f * new_zr * new_zi + y;

        zr2 = zr * zr;
        zi2 = zi * zi;

        iter += 2; // Two iterations per loop
    }

    if (iter < max_iter) {
        float log_zn = logf(zr2 + zi2) / 2.0f;
        float nu = logf(log_zn / logf(2.0f)) / logf(2.0f);
        return iter + 1 - nu;
    } else {
        return max_iter;
    }
}

__global__ void mandelbrot_kernel_manual_unroll(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius_squared, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < width && idy < height) {
        float pixel_size_x = (x_max - x_min) / width;
        float pixel_size_y = (y_max - y_min) / height;
        float real = x_min + idx * pixel_size_x;
        float imag = y_min + idy * pixel_size_y;

        results[idy * width + idx] = mandelbrot_escape_time_manual_unroll(real, imag, max_iter, escape_radius_squared);
    }
}

__global__ void mandelbrot_kernel_smooth_manual_unroll(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius_squared, float* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < width && idy < height) {
        float pixel_size_x = (x_max - x_min) / width;
        float pixel_size_y = (y_max - y_min) / height;
        float real = x_min + idx * pixel_size_x;
        float imag = y_min + idy * pixel_size_y;

        results[idy * width + idx] = mandelbrot_smooth_color_manual_unroll(real, imag, max_iter, escape_radius_squared);
    }
}

extern "C" void compute_grid_cuda_manual_unroll(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius_squared, int* results) {
    int* d_results;
    size_t size = width * height * sizeof(int);
    hipMalloc(&d_results, size);
    
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x, (height + threads_per_block.y - 1) / threads_per_block.y);

    mandelbrot_kernel_manual_unroll<<<num_blocks, threads_per_block>>>(x_min, y_min, x_max, y_max, width, height, max_iter, escape_radius_squared, d_results);

    hipMemcpy(results, d_results, size, hipMemcpyDeviceToHost);
    hipFree(d_results);
}

extern "C" void compute_grid_cuda_smooth_manual_unroll(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius_squared, float* results) {
    float* d_results;
    size_t size = width * height * sizeof(float);
    hipMalloc(&d_results, size);
    
    dim3 threads_per_block(16, 16);
    dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x, (height + threads_per_block.y - 1) / threads_per_block.y);

    mandelbrot_kernel_smooth_manual_unroll<<<num_blocks, threads_per_block>>>(x_min, y_min, x_max, y_max, width, height, max_iter, escape_radius_squared, d_results);

    hipMemcpy(results, d_results, size, hipMemcpyDeviceToHost);
    hipFree(d_results);
}
