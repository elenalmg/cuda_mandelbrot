#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"

namespace py = pybind11;

// Sript qui fonctionne, mais je n'arrive pas à le transformer en module python
// obliger de mettre dans un dossier commun avec le CMakeLists car sinon cela ne trouve pas le script

// ou dans un autre fichier ?
__device__ int _compute_escape_iter(float x, float y, int max_iter, float escape_radius) {
    hipFloatComplex c = make_hipFloatComplex(x, y);
    hipFloatComplex z = make_hipFloatComplex(0.0f, 0.0f);
    for (int i = 0; i < max_iter; i++) {
        z = hipCaddf(hipCmulf(z, z), c);
        if (hipCrealf(z) * hipCrealf(z) + hipCimagf(z) * hipCimagf(z) >= escape_radius * escape_radius) {
            return i;
        }
    }
    return -1;
}

__global__ void compute_grid_kernel(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, int* results, float escape_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    float pixel_size_x = (x_max - x_min) / width;
    float pixel_size_y = (y_max - y_min) / height;
    if (x < width && y < height) {
        float real = x_min + x * pixel_size_x;
        float imag = y_min + y * pixel_size_y;
        results[y * width + x] = _compute_escape_iter(real, imag, max_iter, escape_radius);
    }
}


class MandelbrotCuda {
public:
    float escape_radius;

    MandelbrotCuda(float escape_radius) : escape_radius(escape_radius) {}

    __device__ int compute_escape_iter(float x, float y, int max_iter) {
        return _compute_escape_iter(x, y, max_iter, escape_radius);
    }

    void compute_grid(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter) {
        int* results;
        hipMallocManaged(&results, width * height * sizeof(int));

        dim3 block_size(16, 16);
        dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);
        compute_grid_kernel<<<grid_size, block_size>>>(x_min, y_min, x_max, y_max, width, height, max_iter, results, escape_radius);

        hipDeviceSynchronize();
        
        hipFree(results);
    }
};


void mandelbrot_cuda(float x_min, float y_min, float x_max, float y_max, int width, int height, int max_iter, float escape_radius, py::array_t<int> results) {
    int* results_ptr = static_cast<int*>(results.request().ptr);

    MandelbrotCuda mandelbrot(escape_radius);
    mandelbrot.compute_grid(x_min, y_min, x_max, y_max, width, height, max_iter, results_ptr);
}

PYBIND11_MODULE(mandelbrot, m) {
    m.def("mandelbrot_cuda", &mandelbrot_cuda, "Compute Mandelbrot set using CUDA",
          py::arg("x_min"), py::arg("y_min"), py::arg("x_max"), py::arg("y_max"),
          py::arg("width"), py::arg("height"), py::arg("max_iter"), py::arg("escape_radius"),
          py::arg("results"));
}